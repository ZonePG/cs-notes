
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <time.h>
#include <algorithm>

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])
#define WARP_SIZE 32

void cpuElementWise_add(float *a, float *b, float *c, const int N) {
    for (int i = 0; i < N; i++) {
        c[i] = a[i] + b[i];
    }
}

float testError(
    void (*gpuElementWise_add)(float *, float *, float *, const int),
    dim3 gridDim, dim3 blockDim, const int N) {
    size_t size_a = N * sizeof(float);
    size_t size_b = N * sizeof(float);
    size_t size_c = N * sizeof(float);

    float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c, *h_d_c;
    h_a = (float *)malloc(size_a);
    h_b = (float *)malloc(size_b);
    h_c = (float *)malloc(size_c);
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);
    h_d_c = (float *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < N; i++)
        h_a[i] = rand() / float(RAND_MAX);
    for (int i = 0; i < N; i++)
        h_b[i] = rand() / float(RAND_MAX);
    hipMemset(d_c, 15, size_c);

    cpuElementWise_add(h_a, h_b, h_c, N);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    gpuElementWise_add<<<gridDim, blockDim>>>(d_a, d_b, d_c, N);
    hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost);

    float max_error = 0.0;
    for (int i = 0; i < N; i++) {
        float this_error = abs(h_d_c[i] - h_c[i]);
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = std::max(max_error, this_error);
    }

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_d_c);

    return max_error;
}

float testPerformance(
    void (*gpuElementWise_add)(float *, float *, float *, const int),
    dim3 gridDim, dim3 blockDim, const int N, const int repeat) {
    size_t size_a = N * sizeof(float);
    size_t size_b = N * sizeof(float);
    size_t size_c = N * sizeof(float);

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    for (int i = 0; i < repeat; i++)
        gpuElementWise_add<<<gridDim, blockDim>>>(d_a, d_b, d_c, N);
    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return sec;
}

// ElementWise Add
// grid(N/128), block(128)
// a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add(float *a, float *b, float *c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) c[idx] = a[idx] + b[idx];
}

// ElementWise Add + Vec4
// grid(N/128), block(128/4)
// a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add_vec4(float *a, float *b, float *c, int N) {
    int idx = 4 * (blockIdx.x * blockDim.x + threadIdx.x);
    if (idx < N) {
        float4 reg_a = FLOAT4(a[idx]);
        float4 reg_b = FLOAT4(b[idx]);
        float4 reg_c;
        reg_c.x = reg_a.x + reg_b.x;
        reg_c.y = reg_a.y + reg_b.y;
        reg_c.z = reg_a.z + reg_b.z;
        reg_c.w = reg_a.w + reg_b.w;
        FLOAT4(c[idx]) = reg_c;
    }
}

int main(void) {
    {
        printf("\nKernal = elementwise_add\n");
        const int N = 512;
        dim3 blockDim(128);
        dim3 gridDim(N / 128);
        float max_error = testError(elementwise_add, gridDim, blockDim, N);
        printf("Max Error = %f\n", max_error);
    }

    {
        printf("\nKernal = dot_vec4\n");
        const int N = 512;
        dim3 blockDim(128 / 4);
        dim3 gridDim(N / 128);
        float max_error = testError(elementwise_add_vec4, gridDim, blockDim, N);
        printf("Max Error = %f\n", max_error);
    }

    const int N_list[15] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int TESTNUM = 15;
    const int outer_repeat = 10, inner_repeat = 1;

    for (int i = 0; i < TESTNUM; i++) {
        const int N = N_list[i];

        dim3 blockDim(128 / 4);
        dim3 gridDim(N / 128);

        double max_sec = 0.0;
        double min_sec = DBL_MAX;
        double total_sec = 0.0;

        for (int j = 0; j < outer_repeat; j++) {
            double this_sec = testPerformance(elementwise_add_vec4, gridDim, blockDim, N, inner_repeat);
            max_sec = std::max(max_sec, this_sec);
            min_sec = std::min(min_sec, this_sec);
            total_sec += this_sec;
        }

        double avg_sec = total_sec / outer_repeat;
        double avg_Gflops = ((double)N * 2) / 1024 / 1024 / 1024 / avg_sec;

        printf("N = %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", N, min_sec, avg_sec, max_sec, avg_Gflops);
    }

    return 0;
}